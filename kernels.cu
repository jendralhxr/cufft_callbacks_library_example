#include <cassert>
#include <hip/hip_fp16.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>
#include <stdio.h>

#include "kernels.h"

namespace kernels {

hipfftHandle fft_plan = 0;
void create_fft_plan(const int side_size){
  CheckCudaErrors(hipfftPlan2d(&fft_plan, side_size, side_size, HIPFFT_C2C));
  CheckCudaErrors(hipDeviceSynchronize());
}



void destroy_fft_plan(){
  CheckCudaErrors(hipfftDestroy(fft_plan));
}



__device__ hipfftComplex cu_ifft_prescale(
    void *data_ptr,
    size_t offset,
    void *callerInfo,
    void *sharedPtr)
{
  cfloat element = ((cfloat*)data_ptr)[offset];
  cfloat tf_element = ((cfloat*)callerInfo)[offset];
  return hipCmulf(element,tf_element);
}



__device__ void cu_ifft_post_normalize(
  void *data_ptr,
  size_t offset,
  hipfftComplex element,
  void *callerInfo,
  void *sharedPtr
){
  element.x /= (8*8);
  element.y /= (8*8);
  ((cfloat*)data_ptr)[offset] = element;
}



__device__ hipfftCallbackLoadC  d_loadCallbackPtr  = cu_ifft_prescale;
__device__ hipfftCallbackStoreC d_storeCallbackPtr = cu_ifft_post_normalize;

void scaled_ifft2_inplace(cfloat *const data_dev_ptr, cfloat *const scaling_dev_ptr, const int n, const bool generate_plan){
  hipfftHandle my_fft_plan;

  if(generate_plan){
    hipfftPlan2d(&my_fft_plan, n, n, HIPFFT_C2C);
  } else if(!fft_plan){
    throw std::runtime_error("No FFT plan!");
  } else {
    my_fft_plan = fft_plan;
  }

  hipfftCallbackLoadC h_loadCallbackPtr;
  hipfftCallbackStoreC h_storeCallbackPtr;
  CheckCudaErrors(hipMemcpyFromSymbol(&h_loadCallbackPtr,
                                        HIP_SYMBOL(d_loadCallbackPtr),
                                        sizeof(h_loadCallbackPtr)));
  CheckCudaErrors(hipMemcpyFromSymbol(&h_storeCallbackPtr,
                                        HIP_SYMBOL(d_storeCallbackPtr),
                                        sizeof(h_storeCallbackPtr)));

  CheckCudaErrors(hipfftXtSetCallback(my_fft_plan,
                          (void **)&h_loadCallbackPtr,
                          HIPFFT_CB_LD_COMPLEX,
                          (void**)&scaling_dev_ptr));

  CheckCudaErrors(hipfftXtSetCallback(my_fft_plan,
                              (void **)&h_storeCallbackPtr,
                              HIPFFT_CB_ST_COMPLEX,
                              0));

  CheckCudaErrors(hipDeviceSynchronize());

  CheckCudaErrors(hipfftExecC2C(my_fft_plan, data_dev_ptr, data_dev_ptr, HIPFFT_BACKWARD));

  CheckCudaErrors(hipDeviceSynchronize());

  if(generate_plan){
    CheckCudaErrors(hipfftDestroy(my_fft_plan));
  }
}

}