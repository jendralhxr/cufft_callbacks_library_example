#include <cassert>
#include <hip/hip_fp16.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>
#include <stdio.h>

#include "kernels.h"

namespace kernels {

__device__ hipfftComplex cu_ifft_prescale(
    void *data_ptr,
    size_t offset,
    void *callerInfo,
    void *sharedPtr)
{
  cfloat element = ((cfloat*)data_ptr)[offset];
  cfloat tf_element = ((cfloat*)callerInfo)[offset];
  return hipCmulf(element,tf_element);
}



__device__ void cu_ifft_post_normalize(
  void *data_ptr,
  size_t offset,
  hipfftComplex element,
  void *callerInfo,
  void *sharedPtr
){
  element.x /= (8*8);
  element.y /= (8*8);
  ((cfloat*)data_ptr)[offset] = element;
}



__device__ hipfftCallbackLoadC  d_loadCallbackPtr  = cu_ifft_prescale;
__device__ hipfftCallbackStoreC d_storeCallbackPtr = cu_ifft_post_normalize;

void scaled_ifft2_inplace(cfloat *const data_dev_ptr, cfloat *const scaling_dev_ptr, const int n){
  hipfftHandle fft_plan;
  hipfftPlan2d(&fft_plan, n, n, HIPFFT_C2C);

  hipfftCallbackLoadC h_loadCallbackPtr;
  hipfftCallbackStoreC h_storeCallbackPtr;
  CheckCudaErrors(hipMemcpyFromSymbol(&h_loadCallbackPtr,
                                        HIP_SYMBOL(d_loadCallbackPtr),
                                        sizeof(h_loadCallbackPtr)));
  CheckCudaErrors(hipMemcpyFromSymbol(&h_storeCallbackPtr,
                                        HIP_SYMBOL(d_storeCallbackPtr),
                                        sizeof(h_storeCallbackPtr)));

  CheckCudaErrors(hipfftXtSetCallback(fft_plan,
                          (void **)&h_loadCallbackPtr,
                          HIPFFT_CB_LD_COMPLEX,
                          (void**)&scaling_dev_ptr));

  CheckCudaErrors(hipfftXtSetCallback(fft_plan,
                              (void **)&h_storeCallbackPtr,
                              HIPFFT_CB_ST_COMPLEX,
                              0));

  CheckCudaErrors(hipDeviceSynchronize());

  CheckCudaErrors(hipfftExecC2C(fft_plan, data_dev_ptr, data_dev_ptr, HIPFFT_BACKWARD));

  CheckCudaErrors(hipDeviceSynchronize());
  CheckCudaErrors(hipfftDestroy(fft_plan));
}

}